#include "hip/hip_runtime.h"
/**
*   CS6023: GPU Programming 
*   Assignment 2
*   
*   Please don't change any existing code in this file.
*
*   Please add necessary memory APIs for your implementation. Use hipFree() 
*   to free up memory as soon as you're done with an allocation. 
*   This will ensure that you don't run out of memory while running
*   large test cases. Use the minimum required memory for your 
*   implementation. DO NOT change the kernel configuration parameters.
*/

#include <chrono>
#include <fstream>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

using std::cin;
using std::cout;

typedef long long ll;


//kernel for convolution
__global__ void dkernel(long int* a1, long int* b1, long int* ans, int m, int n, int k) {
    //creating shared memoery 
    extern __shared__ long int s[];

    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned i = id / n;
    unsigned j = id % n;

    // Initialize shared memory for caching input matrices
    if (threadIdx.x == 0) {
        for (long int i = 0; i < k * k; i++) {
            s[i] = b1[i];
        }
    }
    __syncthreads();


       // Compute convolution using cached input matrices in shared memory
    long int sum = 0;
    for (long int a = -k / 2; a <= k / 2; a++) {
        if (i + a < 0 || i + a >= m)
            continue;

        for (long int b = -k / 2; b <= k / 2; b++) {
            if (j + b < 0 || j + b >= n)
                continue;

            sum += a1[(i + a) * n + (b + j)] * s[(a + k / 2) * k + (b + k / 2)];
        }
    }
    //appying coalesing here
    // Writing result to output matrix with coalesced memory access
    ans[id] = sum;
}



int main(int argc, char** argv) {

    int m,n,k;
    cin>>m>>n>>k;


    long int* h_mat = new long int[m * n];
    long int* h_filter = new long int[k * k];

    long int* h_ans = new long int[m * n];


    for (long int i = 0; i < m * n; i++) {
        cin>>h_mat[i];
    }

    for (long int i = 0; i < k * k; i++) {
        cin>>h_filter[i];
    }

    /**
     * 
     * DO NOT CHANGE ANYTHING ABOVE THIS LINE
     * 
    **/

    /****************************************************Start Here***********************************************************/
    
    long int* g_ans;
    long int* g_mat;
    long int* g_filter;
    hipMalloc(&g_mat, sizeof(long int) * m * n);
    hipMalloc(&g_filter, sizeof(long int) * k * k);
    hipMalloc(&g_ans, sizeof(long int) * m * n);
    hipMemcpy(g_mat, h_mat, sizeof(long int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(g_filter, h_filter, sizeof(long int) * k * k, hipMemcpyHostToDevice);

    //kernel launch
    auto start = std::chrono::high_resolution_clock::now();//keep it just before the kernel launch
    dkernel << <m, n, sizeof(long int)* (k * k) >> > (a1, b1, an, m, n, k);
    auto end = std::chrono::high_resolution_clock::now();//keep it just after the kernel launch
    
    //copying the final output to h_ans
    hipMemcpy(h_ans, g_ans, sizeof(long int) * m * n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    hipFree(g_ans);
    hipFree(g_mat);
    hipFree(g_filter);
    
    
    /*$$$$$$$$$$$$$$$$$$$$$$$$Make sure your final output from the device is stored in h_ans.$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$*/
    std::chrono::duration<double> elapsed1 = end - start;
    /**
     * 
     * DO NOT CHANGE ANYTHING BELOW THIS LINE
     * 
    */


    
    std::ofstream file("cuda.out");
    if (file.is_open()) {
        for (long int i = 0; i < m; i++) {
            for (long int j = 0; j < n; j++) {
                file << h_ans[i * n + j] << " ";
            }
            file << "\n";
        }
        file.close();
    } else {
        std::cout << "Unable to open file";
    }

    std::ofstream file2("cuda_timing.out");
    if(file2.is_open()) {
        file2 << elapsed1.count() << "\n";
        file2.close();
    } else {
        std::cout << "Unable to open file";
    }

    return 0;
}